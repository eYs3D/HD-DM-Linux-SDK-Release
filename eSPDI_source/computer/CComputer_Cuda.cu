#include "CComputer_Cuda.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "debug.h"
#include "Cuda_Kernel.h"
#include "objCuda.h"

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      LOGE("GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }


bool CComputer_Cuda::CudaSupport()
{
    return GetCudaDeviceIndex() != -1;
}

int CComputer_Cuda::GetCudaDeviceIndex()
{    
    int count;
    hipError_t error_id = hipGetDeviceCount(&count);
    if (hipSuccess != error_id)
    {
        LOGE("hipGetDeviceCount returned %s\n",
           hipGetErrorString(error_id));
        return -1;
    }

    if (0 == count)
    {
        LOGI("There are no available device(s) that support CUDA\n");
        return -1;
    }

    for (int i = 0; i < count; i++)
    {
        hipDeviceProp_t prop;
        if (hipGetDeviceProperties(&prop, i) == hipSuccess)
        {
            LOGD("CUDA Capability Major/Minor version number:    %d.%d\n",
           prop.major, prop.minor);
            return i;
        }
    }
    
    LOGI("There are no available device(s) that support CUDA\n");
    return -1;      
}


CComputer_Cuda::CComputer_Cuda():
CComputer_Default()
{
    SetType(CUDA);
    int index = GetCudaDeviceIndex();
    if (-1 != index) 
    {
        hipSetDevice(index);
    }
}

CComputer_Cuda::~CComputer_Cuda()
{
    hipDeviceReset();
}

int CComputer_Cuda::FlyingDepthCancellation_D8(unsigned char *pdepthD8, int width, int height)
{
    objCuda cudaObj;
    unsigned char *cuda_LImg = cudaObj.AllocateDeviceMemory(width * height, pdepthD8);
    unsigned char *cuda_DImg = cudaObj.AllocateDeviceMemory(width * height, pdepthD8);
    unsigned char *cuda_out = (unsigned char *)cudaObj.AllocateDeviceMemory(width * height);

    dim3 size(width, height);
    cudaObj.Lanuch(size, Kernel_FlyingDepthCancellation_D8, cuda_LImg, cuda_DImg, cuda_out, width, height);

    cudaObj.CopyCudaDeviceMemoryToHost(pdepthD8, cuda_out, width * height);

    return ETronDI_OK;
}

int CComputer_Cuda::FlyingDepthCancellation_D11(unsigned char *pdepthD11, int width, int height)
{    
    objCuda cudaObj;
    unsigned char *cuda_LImg = cudaObj.AllocateDeviceMemory(width * height * 2, pdepthD11);
    unsigned short *cuda_DImg = (unsigned short *)cudaObj.AllocateDeviceMemory(width * height * 2, pdepthD11);
    unsigned short *cuda_out = (unsigned short *)cudaObj.AllocateDeviceMemory(width * height * 2);

    dim3 size(width, height);
    cudaObj.Lanuch(size, Kernel_FlyingDepthCancellation_D11, cuda_LImg, cuda_DImg, cuda_out, width, height);

    cudaObj.CopyCudaDeviceMemoryToHost(pdepthD11, cuda_out, width * height * 2);

    return ETronDI_OK;
}

int CComputer_Cuda::ColorFormat_to_RGB24(unsigned char *ImgDst, unsigned char *ImgSrc,
                                         int SrcSize, int width, int height,
                                         EtronDIImageType::Value type)
{
    if (!ImgDst || !ImgSrc)
        return ETronDI_NullPtr;
    if (!width || !height)
        return ETronDI_ErrBufLen;
    if (EtronDIImageType::COLOR_YUY2 != type)
        return ETronDI_NotSupport;
    
    objCuda cudaObj;
    unsigned char *cuda_src = cudaObj.AllocateDeviceMemory(width * height * 2, ImgSrc);
    unsigned char *cuda_dst = (unsigned char *)cudaObj.AllocateDeviceMemory(width * height * 3);    

    dim3 size(width / 2, height);
    cudaObj.Lanuch(size, Kernel_YUY2_to_RGB24, cuda_src, cuda_dst, width / 2, height);

    cudaObj.CopyCudaDeviceMemoryToHost(ImgDst, cuda_dst, width * height * 3);

    return ETronDI_OK;
}

int CComputer_Cuda::DepthMerge(unsigned char **pDepthBufList, float *pDepthMergeOut,
                               unsigned char *pDepthMergeFlag,
                               int nDWidth, int nDHeight,
                               float fFocus, float *pBaseline,
                               float *pWRNear, float *pWRFar,
                               float *pWRFusion, int nMergeNum)
{
    objCuda cudaObj;
    
    unsigned char *cuda_depth1 = cudaObj.AllocateDeviceMemory(nDWidth * nDHeight * 2, pDepthBufList[0]);
    unsigned char *cuda_depth2 = cudaObj.AllocateDeviceMemory(nDWidth * nDHeight * 2, pDepthBufList[1]);    
    unsigned char *cuda_depth3 = cudaObj.AllocateDeviceMemory(nDWidth * nDHeight * 2, pDepthBufList[2]);
    float *cuda_depth_merge = cudaObj.AllocateDeviceMemory(sizeof(float) * nDWidth * nDHeight, pDepthMergeOut);
    unsigned char *cuda_flag = cudaObj.AllocateDeviceMemory(nDWidth * nDHeight, pDepthMergeFlag);
    float *cuda_baseline = cudaObj.AllocateDeviceMemory(sizeof(float) * nMergeNum, pBaseline);
    float *cuda_near = cudaObj.AllocateDeviceMemory(sizeof(float) * nMergeNum, pWRNear);
    float *cuda_far = cudaObj.AllocateDeviceMemory(sizeof(float) * nMergeNum, pWRFar);
    float *cuda_fusion = cudaObj.AllocateDeviceMemory(sizeof(float) * nMergeNum, pWRFusion);

    dim3 size(nDWidth, nDHeight);
    cudaObj.Lanuch(size, Kernel_DepthMerge, cuda_depth1, cuda_depth2, cuda_depth3, cuda_depth_merge, cuda_flag, fFocus, cuda_baseline, cuda_near, cuda_far, cuda_fusion,nDWidth, nDHeight);

    cudaObj.CopyCudaDeviceMemoryToHost(pDepthMergeOut, cuda_depth_merge, sizeof(float) * nDWidth * nDHeight);
    cudaObj.CopyCudaDeviceMemoryToHost(pDepthMergeFlag, cuda_flag, nDWidth * nDHeight);

    return ETronDI_OK;
}

int CComputer_Cuda::ImageRotate90(EtronDIImageType::Value imgType, int width, int height,
                                  unsigned char *src, unsigned char *dst, int len, bool clockwise)
{
    auto getBytePerPixel = [=](EtronDIImageType::Value imgType) -> int {
        switch (imgType)
        {
        case EtronDIImageType::DEPTH_8BITS:
            return 1;
        case EtronDIImageType::COLOR_YUY2:
        case EtronDIImageType::DEPTH_8BITS_0x80:
        case EtronDIImageType::DEPTH_11BITS:
        case EtronDIImageType::DEPTH_14BITS:
            return 2;
        case EtronDIImageType::COLOR_RGB24:
            return 3;
        default:
            break;
        }
        return -1;
    };

    unsigned int imgSize = width * height * getBytePerPixel(imgType);
    if (imgSize <= 0 || (int)imgSize > len)
    {
        printf("%s, Image size incorrect = %d, but len = %d\n", __func__, imgSize, len);
        return ETronDI_ErrBufLen;
    }

    if (imgType == EtronDIImageType::COLOR_YUY2)
    {
        objCuda cudaObj;
        unsigned char *cuda_src = cudaObj.AllocateDeviceMemory(imgSize, src);
        unsigned char *cuda_dst = (unsigned char *)cudaObj.AllocateDeviceMemory(imgSize);

        dim3 size(width / 2, height / 2);
        cudaObj.Lanuch(size, rotate_YUY2, cuda_src, cuda_dst, width / 2, height / 2);

        cudaObj.CopyCudaDeviceMemoryToHost(dst, cuda_dst, imgSize);
    }
    else if (imgType == EtronDIImageType::DEPTH_11BITS || imgType == EtronDIImageType::DEPTH_14BITS)
    {
        static const int Depth_dim = 2;

        objCuda cudaObj;
        unsigned char *cuda_src = cudaObj.AllocateDeviceMemory(imgSize, src);
        unsigned char *cuda_dst = (unsigned char *)cudaObj.AllocateDeviceMemory(imgSize);

        dim3 size(width, height);
        cudaObj.Lanuch(size, rotate_dim, cuda_src, cuda_dst, Depth_dim, width, height);

        cudaObj.CopyCudaDeviceMemoryToHost(dst, cuda_dst, imgSize);
    }

    return ETronDI_OK;
}

int CComputer_Cuda::ImageMirro(EtronDIImageType::Value imgType, int width, int height, unsigned char *src, unsigned char *dst)
{
    auto getBytePerPixel = [=](EtronDIImageType::Value imgType) -> int {
        switch (imgType)
        {
        case EtronDIImageType::DEPTH_8BITS:
            return 1;
        case EtronDIImageType::COLOR_YUY2:
        case EtronDIImageType::DEPTH_8BITS_0x80:
        case EtronDIImageType::DEPTH_11BITS:
        case EtronDIImageType::DEPTH_14BITS:
            return 2;
        case EtronDIImageType::COLOR_RGB24:
            return 3;
        default:
            break;
        }
        printf("%s, imgType = %d\n", __func__, imgType);
        return -1;
    };

    unsigned int imgSize = width * height * getBytePerPixel(imgType);
    if (imgSize <= 0)
    {
        printf("%s, Image size incorrect = %d\n", __func__, imgSize);
        return ETronDI_ErrBufLen;
    }

    if (imgType == EtronDIImageType::DEPTH_11BITS || imgType == EtronDIImageType::DEPTH_14BITS)
    {
        static const int Depth_dim = 2;

        objCuda cudaObj;
        unsigned char *cuda_src = cudaObj.AllocateDeviceMemory(imgSize, src);
        unsigned char *cuda_dst = (unsigned char *)cudaObj.AllocateDeviceMemory(imgSize);

        dim3 size(width, height);
        cudaObj.Lanuch(size, mirro_dim, cuda_src, cuda_dst, Depth_dim, width, height);

        cudaObj.CopyCudaDeviceMemoryToHost(dst, cuda_dst, imgSize);
    }

    return ETronDI_OK;
}

int CComputer_Cuda::Resample(const BYTE *ImgSrc, const int SrcW, const int SrcH,
                             BYTE *ImgDst, const int DstW, const int DstH,
                             int BytePerPixel)
{
    objCuda cudaObj;
    unsigned char *cuda_src = (unsigned char *)cudaObj.AllocateDeviceMemory(SrcW * SrcH * BytePerPixel, ImgSrc);
    unsigned char *cuda_dst = (unsigned char *)cudaObj.AllocateDeviceMemory(DstW * DstH * BytePerPixel);

    dim3 size(DstW, DstH);
    cudaObj.Lanuch(size, Kernel_Resample, cuda_src, SrcW, SrcH, cuda_dst, &BytePerPixel, DstW, DstH);

    cudaObj.CopyCudaDeviceMemoryToHost(ImgDst, cuda_dst, DstW * DstH * BytePerPixel);
}

int CComputer_Cuda::GetPointCloud(unsigned char *ImgColor, int CW, int CH,
                                  unsigned char *ImgDepth, int DW, int DH,
                                  PointCloudInfo *pPointCloudInfo,
                                  unsigned char *pPointCloudRGB, float *pPointCloudXYZ, float Near, float Far, unsigned short pid)
{
    BYTE *pDepth = ImgDepth;
    BYTE *pColor = ImgColor;

    int nDstW = DW;
    int nDstH = DH;

    EtronDIImageType::Value imageType = EtronDIImageType::DepthDataTypeToDepthImageType(pPointCloudInfo->wDepthType);

    switch (imageType)
    {
    case EtronDIImageType::DEPTH_11BITS:
    { //d11
        auto Run_PointCloudD11 = [&](auto Kernel) {            
            objCuda cudaObj;
            unsigned char *cuda_color = cudaObj.AllocateDeviceMemory(nDstW * nDstH * 3, pColor);
            unsigned short *cuda_depth = (unsigned short *)cudaObj.AllocateDeviceMemory(nDstW * nDstH * 2, pDepth);
            float *cuda_disparityToW = cudaObj.AllocateDeviceMemory(pPointCloudInfo->disparity_len * sizeof(float), pPointCloudInfo->disparityToW);
            unsigned char *cuda_rgb =  (unsigned char *)cudaObj.AllocateDeviceMemory(nDstW * nDstH * 3);
            float *cuda_xyz =  (float *)cudaObj.AllocateDeviceMemory(nDstW * nDstH * sizeof(float) * 3);

            dim3 size(nDstW, nDstH);
            cudaObj.Lanuch(size, Kernel, cuda_color, cuda_depth, pPointCloudInfo->centerX, pPointCloudInfo->centerY, pPointCloudInfo->focalLength, Near, Far, cuda_disparityToW, cuda_rgb, cuda_xyz, nDstW, nDstH);

            cudaObj.CopyCudaDeviceMemoryToHost(pPointCloudRGB, cuda_rgb, nDstW * nDstH * 3);
            cudaObj.CopyCudaDeviceMemoryToHost(pPointCloudXYZ, cuda_xyz, nDstW * nDstH * sizeof(float) * 3);
        };

        auto Run_PointCloudD11WithK = [&](auto Kernel) {            
            objCuda cudaObj;
            unsigned char *cuda_color = cudaObj.AllocateDeviceMemory(nDstW * nDstH * 3, pColor);
            unsigned short *cuda_depth = (unsigned short *)cudaObj.AllocateDeviceMemory(nDstW * nDstH * 2, pDepth);
            float *cuda_disparityToW = cudaObj.AllocateDeviceMemory(pPointCloudInfo->disparity_len * sizeof(float), pPointCloudInfo->disparityToW);
            unsigned char *cuda_rgb =  (unsigned char *)cudaObj.AllocateDeviceMemory(nDstW * nDstH * 3);
            float *cuda_xyz =  (float *)cudaObj.AllocateDeviceMemory(nDstW * nDstH * sizeof(float) * 3);
            
            dim3 size(nDstW, nDstH);
            cudaObj.Lanuch(size, Kernel, cuda_color, cuda_depth, pPointCloudInfo->centerX, pPointCloudInfo->centerY, pPointCloudInfo->focalLength, pPointCloudInfo->baseline_K, pPointCloudInfo->diff_K, pPointCloudInfo->focalLength_K, Near, Far, cuda_disparityToW, cuda_rgb, cuda_xyz, nDstW, nDstH);

            cudaObj.CopyCudaDeviceMemoryToHost(pPointCloudRGB, cuda_rgb, nDstW * nDstH * 3);
            cudaObj.CopyCudaDeviceMemoryToHost(pPointCloudXYZ, cuda_xyz, nDstW * nDstH * sizeof(float) * 3);

        };
        if (ETronDI_PID_8054 == pid && pPointCloudInfo->focalLength_K)
        {
            Run_PointCloudD11WithK(Kernel_PointCloud_MultipleD11);
        }
        else if (ETronDI_PID_8040S == pid)
        {
            if (pPointCloudInfo->focalLength_K)
                Run_PointCloudD11WithK(Kernel_PointCloud_MultipleCylinderD11);
            else
                Run_PointCloudD11(Kernel_PointCloud_CylinderD11);
        }
        else
        {
            Run_PointCloudD11(Kernel_PointCloud_D11);
        }
    }
    break;
    case EtronDIImageType::DEPTH_14BITS:
    {
        auto Run_PointCloudD14 = [&](auto Kernel) {
            
            objCuda cudaObj;
            unsigned char *cuda_color = cudaObj.AllocateDeviceMemory(nDstW * nDstH * 3, pColor);
            unsigned short *cuda_depth = (unsigned short *)cudaObj.AllocateDeviceMemory(nDstW * nDstH * 2, pDepth);

            unsigned char *cuda_rgb =  (unsigned char *)cudaObj.AllocateDeviceMemory(nDstW * nDstH * 3);
            float *cuda_xyz =  (float *)cudaObj.AllocateDeviceMemory(nDstW * nDstH * sizeof(float) * 3);

            dim3 size(nDstW, nDstH);
            cudaObj.Lanuch(size, Kernel, cuda_color, cuda_depth, pPointCloudInfo->centerX, pPointCloudInfo->centerY, pPointCloudInfo->focalLength, Near, Far, cuda_rgb, cuda_xyz, nDstW, nDstH);

            cudaObj.CopyCudaDeviceMemoryToHost(pPointCloudRGB, cuda_rgb, nDstW * nDstH * 3);
            cudaObj.CopyCudaDeviceMemoryToHost(pPointCloudXYZ, cuda_xyz, nDstW * nDstH * sizeof(float) * 3);

        };

        auto Run_PointCloudD14WidthK = [&](auto Kernel) {
            
            objCuda cudaObj;
            unsigned char *cuda_color = cudaObj.AllocateDeviceMemory(nDstW * nDstH * 3, pColor);
            unsigned short *cuda_depth = (unsigned short *)cudaObj.AllocateDeviceMemory(nDstW * nDstH * 2, pDepth);
            unsigned char *cuda_rgb =  (unsigned char *)cudaObj.AllocateDeviceMemory(nDstW * nDstH * 3);
            float *cuda_xyz =  (float *)cudaObj.AllocateDeviceMemory(nDstW * nDstH * sizeof(float) * 3);
            
            dim3 size(nDstW, nDstH);
            cudaObj.Lanuch(size, Kernel, cuda_color, cuda_depth, pPointCloudInfo->centerX, pPointCloudInfo->centerY, pPointCloudInfo->focalLength, pPointCloudInfo->baseline_K, pPointCloudInfo->diff_K, pPointCloudInfo->focalLength_K, Near, Far, cuda_rgb, cuda_xyz, nDstW, nDstH);

            cudaObj.CopyCudaDeviceMemoryToHost(pPointCloudRGB, cuda_rgb, nDstW * nDstH * 3);
            cudaObj.CopyCudaDeviceMemoryToHost(pPointCloudXYZ, cuda_xyz, nDstW * nDstH * sizeof(float) * 3);
        };
        if (ETronDI_PID_8054 == pid && pPointCloudInfo->focalLength_K)
        {
            Run_PointCloudD14WidthK(Kernel_PointCloud_MultipleD14);
        }
        else if (ETronDI_PID_8040S == pid)
        {
            if (pPointCloudInfo->focalLength_K)
                Run_PointCloudD14WidthK(Kernel_PointCloud_MultipleCylinderD14);
            else
                Run_PointCloudD14(Kernel_PointCloud_CylinderD14);
        }
        else
            Run_PointCloudD14(Kernel_PointCloud_D14);
    }
    break;
    default:
    { //d8
        objCuda cudaObj;
        unsigned char *cuda_color = cudaObj.AllocateDeviceMemory(nDstW * nDstH * 3, pColor);
        unsigned char *cuda_depth = cudaObj.AllocateDeviceMemory(nDstW * nDstH, pDepth);
        float *cuda_disparityToW = cudaObj.AllocateDeviceMemory(pPointCloudInfo->disparity_len * sizeof(float), pPointCloudInfo->disparityToW);
        unsigned char *cuda_rgb =  (unsigned char *)cudaObj.AllocateDeviceMemory(nDstW * nDstH * 3);
        float *cuda_xyz =  (float *)cudaObj.AllocateDeviceMemory(nDstW * nDstH * sizeof(float) * 3);

        dim3 size(nDstW, nDstH);
        cudaObj.Lanuch(size, Kernel_PointCloud_D8, cuda_color, cuda_depth, pPointCloudInfo->centerX, pPointCloudInfo->centerY, pPointCloudInfo->focalLength,Near, Far, cuda_disparityToW, cuda_rgb, cuda_xyz, nDstW, nDstH);

        cudaObj.CopyCudaDeviceMemoryToHost(pPointCloudRGB, cuda_rgb, nDstW * nDstH * 3);
        cudaObj.CopyCudaDeviceMemoryToHost(pPointCloudXYZ, cuda_xyz, nDstW * nDstH * sizeof(float) * 3);
    }
    break;
    }
    return ETronDI_OK;
}

int CComputer_Cuda::TableToData(int width, int height, int TableSize, unsigned short *Table,
                                unsigned short *Src, unsigned short *Dst)
{
    const int DepthSize = width * height * sizeof(unsigned short);
    
    objCuda cudaObj;
    unsigned short *cuda_table = cudaObj.AllocateDeviceMemory(TableSize, Table);
    unsigned short *cuda_src = cudaObj.AllocateDeviceMemory(DepthSize, Src);
    unsigned short *cuda_dst = cudaObj.AllocateDeviceMemory(DepthSize, Dst);

    dim3 size(width, height);
    cudaObj.Lanuch(size, Kernel_TableToData, cuda_table, cuda_src, cuda_dst, width, height);

    cudaObj.CopyCudaDeviceMemoryToHost(Dst, cuda_dst, DepthSize);

    return ETronDI_OK;
}
